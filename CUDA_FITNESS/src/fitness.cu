#include "hip/hip_runtime.h"
#include "../include/fitness.h"
#include "my_utils/Profiler.hpp"
#include <exception>

static unsigned char* x = nullptr;
static std::mutex mxX;
__global__ void fitness_v1_RGBA2(int n, unsigned char* pA, unsigned char* pB)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        // brga to rgba
        float absR = fabsf(pA[i * 4 + 0] - pB[i * 4 + 2]);
        float absG = fabsf(pA[i * 4 + 1] - pB[i * 4 + 1]);
        float absB = fabsf(pA[i * 4 + 2] - pB[i * 4 + 0]);
        float absA = fabsf(pA[i * 4 + 3] - pB[i * 4 + 3]);
#define AlphaMulti 1.f
        float val2 = (absR + absG + absB) + absA * AlphaMulti;
        val2 = (float)(255.f - val2 / (3.f + AlphaMulti)) / 255.f;
        memcpy(&pB[(i)*4], &val2, sizeof(float));
    }
}

// Offloads a lot of addition of scores offset times to gpu
__global__ void calculateFitnessFromArray(int offset, float* arr, size_t size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    i *= offset;
    if (i < size - offset) {
        for (size_t y = 1; y < offset; y++) {
            arr[i] += arr[i + y];
        }
        // arr[i] /= offset;
    }
}
float calculateFitness(unsigned char* img_data, unsigned char* surface_data, int _width, int _height)
{

    unsigned char* test;
    // Allocate Unified Memory – accessible from CPU or GPU
    int size = _width * _height;

    hipMallocManaged(&test, 4 * size * sizeof(unsigned char));

    hipMemcpy(test, surface_data, 4 * size, hipMemcpyDefault);
    float result = calculateFitnessGL(img_data, test, _width, _height);
    // Free memory
    hipFree(test);
    return (result);
}

float calculateFitnessGL(unsigned char* img_data, unsigned char* surface_data, int _width, int _height)
{
    // newTimer("calculateFitnessGL");
    int size = _width * _height;
    mxX.lock();
    if (x == nullptr) {
        hipMallocManaged(&x, 4 * size * sizeof(unsigned char));
        hipMemcpy(x, img_data, 4 * size, hipMemcpyDefault);
    }
    mxX.unlock();
    {
        // newTimer("calculate fitness_v1_RGBA2");
        fitness_v1_RGBA2<<<_width, _height>>>(size, x, surface_data);
        hipError_t ce = hipGetLastError();
        if (ce != hipSuccess) {
            throw std::runtime_error(hipGetErrorString(ce));
        }
        hipDeviceSynchronize();
    }
    // increasing of this number lowers calculate fitness cpu timings 2 times and increases calculateFitnessFromArray 4
    // times, so its important to think about those numbers
    const short offset = 64;
    short rest = size % offset;
    size_t amount = (size - rest) / offset;
    size_t threadAmount = sqrt(amount) + 1;

    {
        // newTimer("calculateFitnessFromArray");
        calculateFitnessFromArray<<<threadAmount, threadAmount>>>(offset, (float*)surface_data, size);
        hipError_t ce = hipGetLastError();
        if (ce != hipSuccess) {
            throw std::runtime_error(hipGetErrorString(ce));
        }
        hipDeviceSynchronize();
    }

    double result = 0;
    float tmp_fitness = 0;
    if (true) {
        {
            newTimer("calculate fitness cpu");
            {
                for (int i = 0; i < amount; i++) {
                    memcpy(&tmp_fitness, (void*)(surface_data + 4 * i * offset), sizeof(float));
                    result += tmp_fitness;
                }
                for (size_t i = 0; i < rest; i++) {
                    memcpy(&tmp_fitness, (void*)(surface_data + amount * offset * 4 + i * 4), sizeof(float));
                    result += tmp_fitness;
                }
                result /= size;
            }
        }
        return (result);
    }
    float tempSurface[size];
    // newTimer("calculate fitness cpu and mem cpy");
    hipMemcpy(tempSurface, surface_data, size * sizeof(float), hipMemcpyDefault);
    {
        {
            for (int i = 0; i < amount; i++) {
                result += tempSurface[i * offset];
            }
            for (size_t i = 0; i < rest; i++) {
                result += tempSurface[amount * offset + i];
            }
            result /= size;
        }
    }
    return (result);
}