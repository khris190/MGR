#include "hip/hip_runtime.h"
#include "../include/fitness.h"
#include "my_utils/Profiler.hpp"

static unsigned char* x = nullptr;
static std::mutex mxX;
__global__ void fitness_v1_RGBA2(int n, unsigned char* pA, unsigned char* pB)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        float absR = fabsf(pA[i * 4 + 0] - pB[i * 4 + 0]);
        float absG = fabsf(pA[i * 4 + 1] - pB[i * 4 + 1]);
        float absB = fabsf(pA[i * 4 + 2] - pB[i * 4 + 2]);
        float absA = fabsf(pA[i * 4 + 3] - pB[i * 4 + 3]);
#define AlphaMulti 1.0f
        float val2 = (absR + absG + absB) + absA * AlphaMulti;
        val2 = (float)(255.f - val2 / (3.f + AlphaMulti)) / 255.f;
        memcpy(&pB[(i)*4], &val2, sizeof(float));
    }
}

// Offloads a lot of addition of scores offset times to gpu
__global__ void calculateFitnessFromArray(int offset, float* arr, size_t size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    i *= offset;
    if (i < size - offset) {
        for (size_t y = 1; y < offset; y++) {
            arr[i] += arr[i + y];
        }
        // arr[i] /= offset;
    }
}
float calculateFitness(unsigned char* img_data, unsigned char* surface_data, int _width, int _height)
{

    unsigned char* test;
    // Allocate Unified Memory – accessible from CPU or GPU
    int size = _width * _height;
    mxX.lock();
    if (x == nullptr) {
        hipMallocManaged(&x, 4 * size * sizeof(unsigned char));
        hipMemcpy(x, img_data, 4 * size, hipMemcpyDefault);
    }
    mxX.unlock();
    hipMallocManaged(&test, 4 * size * sizeof(unsigned char));

    hipMemcpy(test, surface_data, 4 * size, hipMemcpyDefault);

    {
        // newTimer("calculate fitness_v1_RGBA2");
        fitness_v1_RGBA2<<<_width, _height>>>(size, x, test);
        hipError_t ce = hipGetLastError();
        hipDeviceSynchronize();
    }
    // increasing of this number lowers calculate fitness cpu timings 2 times and increases calculateFitnessFromArray 4
    // times, so its important to think about those numbers
    const short offset = 64;
    short rest = size % offset;
    size_t amount = (size - rest) / offset;
    size_t threadAmount = sqrt(amount) + 1;

    {
        // newTimer("calculateFitnessFromArray");
        calculateFitnessFromArray<<<threadAmount, threadAmount>>>(offset, (float*)test, size);
        hipError_t ce = hipGetLastError();
        hipDeviceSynchronize();
    }

    double result = 0;
    float tmp_fitness = 0;
    {
        // newTimer("calculate fitness cpu");
        {
            for (int i = 0; i < amount; i++) {
                memcpy(&tmp_fitness, (void*)(test + 4 * i * offset), sizeof(float));
                result += tmp_fitness;
            }
            for (size_t i = 0; i < rest; i++) {
                memcpy(&tmp_fitness, (void*)(test + amount * offset * 4 + i * 4), sizeof(float));
                result += tmp_fitness;
            }
            result /= size;
        }
    }

    // Free memory
    hipFree(test);
    return (result);
}